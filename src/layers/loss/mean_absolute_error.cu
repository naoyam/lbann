#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_MEAN_ABSOLUTE_ERROR_LAYER_INSTANTIATE
#include "lbann/layers/loss/mean_absolute_error.hpp"

namespace lbann {

namespace {

template <int block_size>
__global__ void fp_kernel(int global_height,
                          int local_height, int local_width,
                          const DataType* __restrict__ prediction,
                          int prediction_ldim,
                          const DataType* __restrict__ ground_truth,
                          int ground_truth_ldim,
                          DataType* __restrict__ contribution) {

  // Indices
  const int tid = threadIdx.x;
  const int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const int bidy = blockIdx.y;
  const int nthreadsx = blockDim.x * gridDim.x;

  // Compute local contribution for each matrix column
  for (int col = bidy; col < local_width; col += gridDim.y) {

    // Compute contributions for each thread
    DataType private_contribution = DataType(0);
    for (int row = gidx; row < local_height; row += nthreadsx) {
      const auto& x = prediction[row + col * prediction_ldim];
      const auto& xhat = ground_truth[row + col * ground_truth_ldim];
      private_contribution += cuda::abs(x - xhat);
    }

    // Shared memory reduction to get contribution for each block
    /// @todo unroll loops
    __shared__ DataType shared_contribution[block_size];
    shared_contribution[tid] = private_contribution;
    for (int stride = block_size / 2; stride > 0; stride /= 2) {
      __syncthreads();
      if (tid < stride) {
        shared_contribution[tid] += shared_contribution[tid + stride];
      }
    }
    if (tid == 0) {
      shared_contribution[0] /= global_height;
      cuda::atomic_add(&contribution[col], shared_contribution[0]);
    }

  }

}

void local_fp_gpu(El::Int height,
                  const AbsMat& local_prediction,
                  const AbsMat& local_ground_truth,
                  AbsMat& local_contribution) {
  El::Zero(local_contribution);
  const auto& local_height = local_prediction.Height();
  const auto& local_width = local_prediction.Width();
  if (local_height > 0 && local_width > 0) {
    const int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
    fp_kernel<block_size>
      <<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
        height, local_height, local_width,
        local_prediction.LockedBuffer(), local_prediction.LDim(),
        local_ground_truth.LockedBuffer(), local_ground_truth.LDim(),
        local_contribution.Buffer());
  }
}

template <int block_size>
__global__ void bp_kernel(int global_height,
                          int local_height, int local_width,
                          const DataType* __restrict__ prediction,
                          int prediction_ldim,
                          const DataType* __restrict__ ground_truth,
                          int ground_truth_ldim,
                          const DataType* __restrict__ gradient_wrt_output,
                          DataType* __restrict__ gradient_wrt_prediction,
                          int gradient_wrt_prediction_ldim,
                          DataType* __restrict__ gradient_wrt_ground_truth,
                          int gradient_wrt_ground_truth_ldim) {

  // Indices
  const int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const int bidy = blockIdx.y;
  const int nthreadsx = blockDim.x * gridDim.x;

  // Compute gradients
  for (int col = bidy; col < local_width; col += gridDim.y) {
    const auto& dy = gradient_wrt_output[col];
    for (int row = gidx; row < local_height; row += nthreadsx) {
      const auto& x = prediction[row + col * prediction_ldim];
      const auto& xhat = ground_truth[row + col * ground_truth_ldim];
      auto& dx = gradient_wrt_prediction[row + col * gradient_wrt_prediction_ldim];
      auto& dxhat = gradient_wrt_ground_truth[row + col * gradient_wrt_ground_truth_ldim];
      if (x > xhat) {
        dx = dy / global_height;
        dxhat = -dy / global_height;
      } else if (x < xhat) {
        dx = -dy / global_height;
        dxhat = dy / global_height;
      } else {
        dx = DataType(0);
        dxhat = DataType(0);
      }
    }
  }

}

void local_bp_gpu(El::Int height,
                  const AbsMat& local_prediction,
                  const AbsMat& local_ground_truth,
                  const AbsMat& local_gradient_wrt_output,
                  AbsMat& local_gradient_wrt_prediction,
                  AbsMat& local_gradient_wrt_ground_truth) {
  const auto& local_height = local_prediction.Height();
  const auto& local_width = local_prediction.Width();
  if (local_height > 0 && local_width > 0) {
    const int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
    bp_kernel<block_size>
      <<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
        height, local_height, local_width,
        local_prediction.LockedBuffer(), local_prediction.LDim(),
        local_ground_truth.LockedBuffer(), local_ground_truth.LDim(),
        local_gradient_wrt_output.LockedBuffer(),
        local_gradient_wrt_prediction.Buffer(),
        local_gradient_wrt_prediction.LDim(),
        local_gradient_wrt_ground_truth.Buffer(),
        local_gradient_wrt_ground_truth.LDim());
  }
}

} // namespace

template <>
void mean_absolute_error_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>
     ::local_fp_compute(El::Int height,
                        const AbsMat& local_prediction,
                        const AbsMat& local_ground_truth,
                        AbsMat& local_contribution) {
  local_fp_gpu(height, local_prediction, local_ground_truth,
               local_contribution);
}

template <>
void mean_absolute_error_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>
     ::local_bp_compute(El::Int height,
                        const AbsMat& local_prediction,
                        const AbsMat& local_ground_truth,
                        const AbsMat& local_gradient_wrt_output,
                        AbsMat& local_gradient_wrt_prediction,
                        AbsMat& local_gradient_wrt_ground_truth) {
  local_bp_gpu(height,
               local_prediction,
               local_ground_truth,
               local_gradient_wrt_output,
               local_gradient_wrt_prediction,
               local_gradient_wrt_ground_truth);
}

template <>
void mean_absolute_error_layer<data_layout::DATA_PARALLEL, El::Device::GPU>
     ::local_fp_compute(El::Int height,
                        const AbsMat& local_prediction,
                        const AbsMat& local_ground_truth,
                        AbsMat& local_contribution) {
  local_fp_gpu(height, local_prediction, local_ground_truth,
               local_contribution);
}

template <>
void mean_absolute_error_layer<data_layout::DATA_PARALLEL, El::Device::GPU>
     ::local_bp_compute(El::Int height,
                        const AbsMat& local_prediction,
                        const AbsMat& local_ground_truth,
                        const AbsMat& local_gradient_wrt_output,
                        AbsMat& local_gradient_wrt_prediction,
                        AbsMat& local_gradient_wrt_ground_truth) {
  local_bp_gpu(height,
               local_prediction,
               local_ground_truth,
               local_gradient_wrt_output,
               local_gradient_wrt_prediction,
               local_gradient_wrt_ground_truth);
}

template class mean_absolute_error_layer<
  data_layout::DATA_PARALLEL, El::Device::GPU>;
template class mean_absolute_error_layer<
  data_layout::MODEL_PARALLEL, El::Device::GPU>;

} // namespace lbann
