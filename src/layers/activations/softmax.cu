#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_SOFTMAX_LAYER_INSTANTIATE
#include "lbann/layers/activations/softmax.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
/** Functor to ensure values are above threshold value */
struct threshold_op {
  __forceinline__ __device__ DataType operator()(const DataType& y) const {
    return cuda::max(y, cuda::sqrt(cuda::min<DataType>()));
  }
};
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD

/** @brief Max functor */
template <class T>
struct max_op {
  __device__ __forceinline__
  DataType operator()(const T& x1, const T& x2) const {
    return cuda::max(x1, x2);
  }
};

/** @brief Kernel for max reduction on matrix columns
 *
 *  Each CUDA block computes the max over a subset of matrix entries
 *  and outputs the result. This is repeated multiple times for
 *  column-wise max reduction.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param values       (height x width) matrix
 *  @param max_values   (nblocksx x width) matrix
 */
template <size_t bsize>
__global__ void reduce_max_kernel(size_t height,
                                  size_t width,
                                  const DataType* __restrict__ values,
                                  size_t values_ldim,
                                  DataType* __restrict__ max_values) {

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidx = blockIdx.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksx = gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {

    // Find largest value for each thread
    DataType thread_max_val{-cuda::infinity<DataType>()};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& val = values[row+col*values_ldim];
      thread_max_val = cuda::max(thread_max_val, val);
    }

    // Find largest value for each block
    const DataType block_max_val
      = cuda::block_reduce<bsize,1,1,DataType,max_op<DataType>>(thread_max_val);
    if (tid == 0) {
      max_values[bidx+col*nblocksx] = block_max_val;
    }

  }

}

/** @brief Compute exp(x-shift)
 *
 *  Also compute sum(exp(x-shift)) for each matrix column.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 */
template <size_t bsize>
__global__ void fp_exp_kernel(size_t height,
                              size_t width,
                              const DataType* __restrict__ input,
                              size_t input_ldim,
                              DataType* __restrict__ output,
                              size_t output_ldim,
                              const DataType* __restrict__ shifts,
                              DataType* __restrict__ sums) {

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {
    const auto& shift = shifts[col];

    // Exponentiate inputs and compute sum for each thread
    DataType thread_sum{0};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& x = input[row+col*input_ldim];
      auto& y = output[row+col*output_ldim];
      y = cuda::exp(x-shift);
      thread_sum += y;
    }

    // Compute sum for each block
    const DataType block_sum = cuda::block_reduce<bsize,1,1>(thread_sum);
    if (tid == 0) {
      cuda::atomic_add(&sums[col], block_sum);
    }

  }

}

/** @brief Compute layer output
 *
 *  y = exp(x-shift) / sum(exp(x-shift))
 *
 *  If @c LBANN_ENABLE_SOFTMAX_THRESHOLD is set, small values are
 *  thresholded to a minimum value to avoid denormalized floats.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param output   On input, constains exp(x-shift). On output,
 *                  contains the layer output.
 *  @param sums     sum(exp(x-shift)) for each column
 */
__global__ void fp_output_kernel(size_t height,
                                 size_t width,
                                 DataType* __restrict__ output,
                                 size_t output_ldim,
                                 const DataType* __restrict__ sums) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t col = gidy; col < width; col += nthreadsy) {
    const auto& denom = sums[col];
    for (size_t row = gidx; row < height; row += nthreadsx) {
      auto& y = output[row+col*output_ldim];
      y /= denom;
#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
      y = cuda::max(y, cuda::sqrt(cuda::min<DataType>()));
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD
    }
  }
}

/** @brief Compute dot(y,dy) for each matrix column
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 */
template <size_t bsize>
__global__ void bp_dot_product_kernel(size_t height,
                                      size_t width,
                                      const DataType* __restrict__ output,
                                      size_t output_ldim,
                                      const DataType* __restrict__ gradient_wrt_output,
                                      size_t gradient_wrt_output_ldim,
                                      DataType* __restrict__ dot_products) {

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {

    // Compute dot product contribution for each thread
    DataType thread_dot_product{0};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& y = output[row+col*output_ldim];
      const auto& dy = gradient_wrt_output[row+col*gradient_wrt_output_ldim];
      thread_dot_product += y * dy;
    }

    // Compute dot product contribution for each block
    const DataType block_dot_product
      = cuda::block_reduce<bsize,1,1>(thread_dot_product);
    if (tid == 0) {
      cuda::atomic_add(&dot_products[col], block_dot_product);
    }

  }

}

/** @brief Compute gradient w.r.t. input
 *
 *  dx = y * (dy - dot(y,dy))
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param dot_products dot(y,dy) for each matrix column
 */
template <size_t bsize>
__global__ void bp_kernel(size_t height,
                          size_t width,
                          const DataType* __restrict__ output,
                          size_t output_ldim,
                          const DataType* __restrict__ gradient_wrt_output,
                          size_t gradient_wrt_output_ldim,
                          const DataType* __restrict__ dot_products,
                          DataType* __restrict__ gradient_wrt_input,
                          size_t gradient_wrt_input_ldim) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t col = gidy; col < width; col += nthreadsy) {
    const auto& y_dot_dy = dot_products[col];
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& y = output[row+col*output_ldim];
      const auto& dy = gradient_wrt_output[row+col*gradient_wrt_output_ldim];
      auto& dx = gradient_wrt_input[row+col*gradient_wrt_input_ldim];
      dx = y * (dy - y_dot_dy);
    }
  }
}

} // namespace

#ifdef LBANN_HAS_DISTCONV
template <>
void softmax_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::
fp_compute_distconv() {
  dc::MPIPrintStreamDebug() << get_name() << ": " << __FUNCTION__;
  assert_always(distconv_enabled());
  m_softmax->forward(m_prev_activations_t, m_activations_t);
  copy_out_activations();
}

template <>
void softmax_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::
bp_compute_distconv() {
  dc::MPIPrintStreamDebug() << get_name() << ": " << __FUNCTION__;
  assert_always(distconv_enabled());
  m_softmax->backward(m_activations_t, m_prev_error_signals_t,
                      m_error_signals_t);
  copy_out_error_signals();
}
#endif // LBANN_HAS_DISTCONV

template <>
void softmax_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::fp_compute() {
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled()) {
    fp_compute_distconv();
    if (!early_terminate_last_iteration()) {
      return;
    }
    // fall through the normal code path to obtain reference results
  }
#endif
  constexpr DataType zero = 0;
  constexpr DataType one = 1;
  const auto& local_input = dynamic_cast<const GPUMat&>(get_local_prev_activations());
  auto& local_output = dynamic_cast<GPUMat&>(get_local_activations());

  hipdnnSoftmaxMode_t cudnn_softmax_mode;
  switch(m_mode) {
    case softmax_mode::INSTANCE:
      cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_INSTANCE;
      break;
    case softmax_mode::CHANNEL:
      cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_CHANNEL;
      break;
    default:
      LBANN_ERROR("Unsupported softmax mode");
  }

  if (!local_input.IsEmpty()) {
    CHECK_CUDNN(hipdnnSoftmaxForward(cudnn::get_handle(),
                                    HIPDNN_SOFTMAX_ACCURATE,
                                    cudnn_softmax_mode,
                                    &one,
                                    m_tensors_cudnn_desc.get_prev_activations(),
                                    local_input.LockedBuffer(),
                                    &zero,
                                    m_tensors_cudnn_desc.get_activations(),
                                    local_output.Buffer()));
#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
    cuda::apply_entrywise_unary_operator<threshold_op>(local_output,
                                                       local_output);
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD
  }
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled() && early_terminate_last_iteration() &&
      keep_original()) {
    dump_reference_activations();
  }
#endif // LBANN_HAS_DISTCONV
}

template <>
void softmax_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::bp_compute() {
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled()) {
    bp_compute_distconv();
    if (!early_terminate_last_iteration()) {
      return;
    }
  }
#endif // LBANN_HAS_DISTCONV
  constexpr DataType zero = 0;
  constexpr DataType one = 1;
  const auto& local_output = dynamic_cast<const GPUMat&>(get_local_activations());
  const auto& local_gradient_wrt_output = dynamic_cast<const GPUMat&>(get_local_prev_error_signals());
  auto& local_gradient_wrt_input = dynamic_cast<GPUMat&>(get_local_error_signals());

  hipdnnSoftmaxMode_t cudnn_softmax_mode;
  switch(m_mode) {
    case softmax_mode::INSTANCE:
      cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_INSTANCE;
      break;
    case softmax_mode::CHANNEL:
      cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_CHANNEL;
      break;
    default:
      LBANN_ERROR("Unsupported softmax mode");
  }

  if (!local_output.IsEmpty()) {
    CHECK_CUDNN(hipdnnSoftmaxBackward(cudnn::get_handle(),
                                     HIPDNN_SOFTMAX_ACCURATE,
                                     cudnn_softmax_mode,
                                     &one,
                                     m_tensors_cudnn_desc.get_activations(),
                                     local_output.LockedBuffer(),
                                     m_tensors_cudnn_desc.get_prev_error_signals(),
                                     local_gradient_wrt_output.LockedBuffer(),
                                     &zero,
                                     m_tensors_cudnn_desc.get_error_signals(),
                                     local_gradient_wrt_input.Buffer()));
  }
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled() && early_terminate_last_iteration() &&
      keep_original()) {
    dump_reference_error_signals();
  }
#endif // LBANN_HAS_DISTCONV
}

template <>
void softmax_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>::fp_compute() {

  if(m_mode != softmax_mode::INSTANCE) {
    LBANN_ERROR("Unsupported softmax mode");
  }

  // Local matrices
  const auto& local_input = dynamic_cast<const GPUMat&>(get_local_prev_activations());
  auto& local_output = dynamic_cast<GPUMat&>(get_local_activations());
  auto& local_workspace = dynamic_cast<GPUMat&>(m_workspace->Matrix());
  const size_t local_height = local_input.Height();
  const size_t local_width = local_input.Width();

  // GPU objects
  auto&& stream = El::GPUManager::Stream();
  auto&& event = El::GPUManager::Event();
  El::SyncInfo<El::Device::GPU> sync_info{stream, event};

  // Find max value in each column
  cuda::thrust::vector<DataType> max_vals;
  if (local_output.IsEmpty()) {
    max_vals.resize(local_width,
                    -std::numeric_limits<DataType>::infinity());
  }
  else {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    max_vals.resize(grid_dims.x * local_width);
    reduce_max_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_input.LockedBuffer(), local_input.LDim(),
      max_vals.data().get());
    while (grid_dims.x > 1) {
      const size_t prev_height = grid_dims.x;
      grid_dims.x = (prev_height + block_size - 1) / block_size;
      cuda::thrust::vector<DataType> prev_vals(std::move(max_vals));
      max_vals.resize(grid_dims.x * local_width);
      reduce_max_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
        prev_height, local_width,
        prev_vals.data().get(), prev_height,
        max_vals.data().get());
    }
  }
  El::mpi::AllReduce(max_vals.data().get(), max_vals.size(),
                     El::mpi::MAX, m_workspace->RedundantComm(),
                     sync_info);

  // Compute exp(x-max_val) and sum(exp(x-max_val))
  El::Zero(*m_workspace);
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    fp_exp_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_input.LockedBuffer(), local_input.LDim(),
      local_output.Buffer(), local_output.LDim(),
      max_vals.data().get(),
      local_workspace.Buffer());
  }
  El::AllReduce(*m_workspace, m_workspace->RedundantComm());

  // Compute output
  // Note: y = exp(x-max_val) / sum(exp(x-max_val))
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    fp_output_kernel<<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_output.Buffer(), local_output.LDim(),
      local_workspace.LockedBuffer());
  }

}

template <>
void softmax_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>::bp_compute() {

  if(m_mode != softmax_mode::INSTANCE) {
    LBANN_ERROR("Unsupported softmax mode");
  }

  // Local matrices
  const auto& local_output = dynamic_cast<const GPUMat&>(get_local_activations());
  const auto& local_gradient_wrt_output = dynamic_cast<const GPUMat&>(get_local_prev_error_signals());
  auto& local_gradient_wrt_input = dynamic_cast<GPUMat&>(get_local_error_signals());
  auto& local_workspace = dynamic_cast<GPUMat&>(m_workspace->Matrix());
  const size_t local_height = local_output.Height();
  const size_t local_width = local_output.Width();

  // GPU objects
  auto&& stream = El::GPUManager::Stream();
  auto&& event = El::GPUManager::Event();
  El::SyncInfo<El::Device::GPU> sync_info{stream, event};

  // Compute dot(y,dy)
  El::Zero(local_workspace);
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    bp_dot_product_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        local_height, local_width,
        local_output.LockedBuffer(),
        local_output.LDim(),
        local_gradient_wrt_output.LockedBuffer(),
        local_gradient_wrt_output.LDim(),
        local_workspace.Buffer());
  }
  El::AllReduce(*m_workspace, m_workspace->RedundantComm());

  // Compute gradient w.r.t. input
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    bp_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_output.LockedBuffer(),
      local_output.LDim(),
      local_gradient_wrt_output.LockedBuffer(),
      local_gradient_wrt_output.LDim(),
      local_workspace.Buffer(),
      local_gradient_wrt_input.Buffer(),
      local_gradient_wrt_input.LDim());
  }

}

// Template instantiation
template class softmax_layer<
  data_layout::DATA_PARALLEL, El::Device::GPU>;
template class softmax_layer<
  data_layout::MODEL_PARALLEL, El::Device::GPU>;

} // namespace lbann
