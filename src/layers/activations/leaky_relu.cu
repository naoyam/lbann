#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_LEAKY_RELU_LAYER_INSTANTIATE
#include "lbann/layers/activations/leaky_relu.hpp"

namespace lbann {

namespace {

/** CUDA kernel for forward prop computation. */
__global__ void fp_kernel(DataType negative_slope,
                          El::Int height,
                          El::Int width,
                          const DataType* __restrict__ input,
                          El::Int input_ldim,
                          DataType* __restrict__ output,
                          El::Int output_ldim) {
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int size = height * width;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int pos = gid; pos < size; pos += num_threads) {
    const auto& row = pos % height;
    const auto& col = pos / height;
    const auto& x = input[row + col * input_ldim];
    auto& y = output[row + col * output_ldim];
    y = (x > DataType(0)) ? x : negative_slope * x;
  }
}

/** CUDA kernel for backprop computation. */
__global__ void bp_kernel(DataType negative_slope,
                          El::Int height,
                          El::Int width,
                          const DataType* __restrict__ input,
                          El::Int input_ldim,
                          const DataType* __restrict__ gradient_wrt_output,
                          El::Int gradient_wrt_output_ldim,
                          DataType* __restrict__ gradient_wrt_input,
                          El::Int gradient_wrt_input_ldim) {
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int size = height * width;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int pos = gid; pos < size; pos += num_threads) {
    const auto& row = pos % height;
    const auto& col = pos / height;
    const auto& x = input[row + col * input_ldim];
    const auto& dy = gradient_wrt_output[row + col * gradient_wrt_output_ldim];
    auto& dx = gradient_wrt_input[row + col * gradient_wrt_input_ldim];
    dx = (x > DataType(0)) ? dy : dy * negative_slope;
  }
}

/** Local forward prop computation. */
void local_fp(DataType negative_slope,
              const AbsMat& input,
              AbsMat& output) {

  // Get CUDA grid dimensions
  // Note: Maximum CUDA grid dimension is 2^32-1
  // (https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications).
  const El::Int height = input.Height();
  const El::Int width = input.Width();
  const El::Int block_dim = 256;
  El::Int grid_dim = (height * width + block_dim - 1) / block_dim;
  if (sizeof(El::Int) > sizeof(unsigned int)
      && grid_dim > std::numeric_limits<uint32_t>::max()) {
    grid_dim = std::numeric_limits<uint32_t>::max();
  }

  // Launch CUDA kernel
  if (grid_dim > 0) {
    fp_kernel<<<grid_dim, block_dim, 0, El::GPUManager::Stream()>>>(
      negative_slope, height, width,
      input.LockedBuffer(), input.LDim(),
      output.Buffer(), output.LDim());
  }

}

/** Local backprop computation. */
void local_bp(DataType negative_slope,
              const AbsMat& input,
              const AbsMat& gradient_wrt_output,
              AbsMat& gradient_wrt_input) {

  // Get CUDA grid dimensions
  // Note: Maximum CUDA grid dimension is 2^32-1
  // (https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications).
  const El::Int height = input.Height();
  const El::Int width = input.Width();
  const El::Int block_dim = 256;
  El::Int grid_dim = (height * width + block_dim - 1) / block_dim;
  if (sizeof(El::Int) > sizeof(unsigned int)
      && grid_dim > std::numeric_limits<uint32_t>::max()) {
    grid_dim = std::numeric_limits<uint32_t>::max();
  }

  // Launch CUDA kernel
  if (grid_dim > 0) {
    bp_kernel<<<grid_dim, block_dim, 0, El::GPUManager::Stream()>>>(
      negative_slope, height, width,
      input.LockedBuffer(), input.LDim(),
      gradient_wrt_output.LockedBuffer(), gradient_wrt_output.LDim(),
      gradient_wrt_input.Buffer(), gradient_wrt_input.LDim());
  }

}

} // namespace

template <>
void leaky_relu_layer<data_layout::DATA_PARALLEL, El::Device::GPU>
       ::fp_compute() {
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled()) {
    fp_compute_distconv();
    if (!early_terminate_last_iteration()) {
      return;
    }
    // fall through the normal code path to obtain reference results
  }
#endif
  local_fp(m_negative_slope,
           get_local_prev_activations(),
           get_local_activations());
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled() && early_terminate_last_iteration() &&
      keep_original()) {
    dump_reference_activations();
  }
#endif // LBANN_HAS_DISTCONV
}
template <>
void leaky_relu_layer<data_layout::DATA_PARALLEL, El::Device::GPU>
     ::bp_compute() {
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled()) {
    bp_compute_distconv();
    if (!early_terminate_last_iteration()) {
      return;
    }
  }
#endif // LBANN_HAS_DISTCONV
  local_bp(m_negative_slope,
           get_local_prev_activations(),
           get_local_prev_error_signals(),
           get_local_error_signals());
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled() && early_terminate_last_iteration() &&
      keep_original()) {
    dump_reference_error_signals();
  }
#endif // LBANN_HAS_DISTCONV
}
template <>
void leaky_relu_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>
       ::fp_compute() {
  local_fp(m_negative_slope,
           get_local_prev_activations(),
           get_local_activations());
}
template <>
void leaky_relu_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>
     ::bp_compute() {
  local_bp(m_negative_slope,
           get_local_prev_activations(),
           get_local_prev_error_signals(),
           get_local_error_signals());
}

template class leaky_relu_layer<
  data_layout::DATA_PARALLEL, El::Device::GPU>;
template class leaky_relu_layer<
  data_layout::MODEL_PARALLEL, El::Device::GPU>;

} // namespace lbann
