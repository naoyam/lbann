#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_BATCH_NORMALIZATION_LAYER_INSTANTIATE
#include "lbann/layers/regularizers/batch_normalization.hpp"
#include "lbann/utils/cuda.hpp"
#include "lbann/execution_contexts/sgd_execution_context.hpp"

namespace lbann {

namespace {

/** CUDA kernel to compute channel sums.
 *  Sums and squares of sums are used to compute mean and variance.
 */
template <El::Int block_size>
__global__ void channel_sums_kernel(
  El::Int channel_height,
  El::Int width,
  const DataType * __restrict__ data, El::Int data_ldim,
        DataType * __restrict__ sums,
        DataType * __restrict__ sqsums) {

  // Indices
  const El::Int tid = threadIdx.x;
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Initialize shared memory
  __shared__ DataType shared_sums[block_size];
  __shared__ DataType shared_sqsums[block_size];

  // Compute row sums in shared memory
  DataType private_sum = 0;
  DataType private_sqsum = 0;
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < width; ++col) {
      const auto& x = data[row + col * data_ldim];
      private_sum += x;
      private_sqsum += x * x;
    }
  }
  shared_sums[tid] = private_sum;
  shared_sqsums[tid] = private_sqsum;

  // Compute channel sum with shared memory reduction
  /// @todo unroll loops
  for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
    __syncthreads();
    if(tid < stride) {
      shared_sums[tid] += shared_sums[tid + stride];
      shared_sqsums[tid] += shared_sqsums[tid + stride];
    }
  }

  // Output channel sum to global memory
  if (tid == 0) {
    cuda::atomic_add(&sums[bidy], shared_sums[0]);
    cuda::atomic_add(&sqsums[bidy], shared_sqsums[0]);
  }

}

/** CUDA kernel to compute statistics.
 *  On input, global_mean and global_var are assumed to contain sums
 *  and squares of sums, respectively.
 */
__global__ void compute_statistics_kernel(
  El::Int num_sums,
  El::Int num_per_sum,
  DataType epsilon,
  DataType decay,
  DataType * __restrict__ global_mean,
  DataType * __restrict__ global_var,
  DataType * __restrict__ global_running_mean,
  DataType * __restrict__ global_running_var) {
  constexpr DataType one = 1;
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int i = gid; i < num_sums; i += num_threads) {

    // Compute mean and variance
    const auto& mean = global_mean[i] / num_per_sum;
    const auto& sqmean = global_var[i] / num_per_sum;
    auto var = num_per_sum * (sqmean - mean * mean) / (num_per_sum - 1);
    var = var > epsilon ? var : epsilon;
    global_mean[gid] = mean;
    global_var[gid] = var;

    // Compute running statistics
    auto& running_mean = global_running_mean[gid];
    auto& running_var = global_running_var[gid];
    running_mean = decay * running_mean + (one - decay) * mean;
    running_var = decay * running_var + (one - decay) * var;

  }

}

/** CUDA kernel to apply batch normalization. */
template <El::Int block_size>
__global__ void batch_normalization_kernel(
  El::Int channel_height,
  El::Int width,
  const DataType * __restrict__ global_input, El::Int input_ldim,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
  const DataType * __restrict__ global_bias,
        DataType * __restrict__ global_output, El::Int output_ldim) {

  // Indices
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];
  const auto& bias = global_bias[bidy];

  // Get reciprocal of standard deviation
  const auto& inv_stdev = cuda::rsqrt(var + epsilon);

  // Apply batch normalization
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& xhat = (x - mean) * inv_stdev;
      const auto& y = scale * xhat + bias;
      global_output[row + col * output_ldim] = y;
    }
  }

}

/** CUDA kernel to compute gradients w.r.t. batch norm parameters. */
template <El::Int block_size>
__global__ void backprop1_kernel(
  El::Int channel_height,
  El::Int width,
  const DataType * __restrict__ global_input,
  El::Int input_ldim,
  const DataType * __restrict__ global_gradient_wrt_output,
  El::Int gradient_wrt_output_ldim,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
        DataType * __restrict__ global_dscale,
        DataType * __restrict__ global_dbias,
        DataType * __restrict__ global_dmean,
        DataType * __restrict__ global_dvar) {

  // Indices
  const El::Int tid = threadIdx.x;
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Initialize shared memory
  __shared__ DataType shared_dscale[block_size];
  __shared__ DataType shared_dbias[block_size];
  __shared__ DataType shared_dmean[block_size];
  __shared__ DataType shared_dvar[block_size];

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];

  // Compute useful constants
  constexpr DataType zero = 0;
  const auto& inv_stdev = cuda::rsqrt(var + epsilon);
  const auto& dvar_factor = inv_stdev * inv_stdev * inv_stdev / 2;

  // Compute row-wise gradient contributions in shared memory
  auto dscale = zero;
  auto dbias = zero;
  auto dmean = zero;
  auto dvar = zero;
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for(El::Int col = 0; col < width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& xhat = (x - mean) * inv_stdev;
      const auto& dy = global_gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      dscale += dy * xhat;
      dbias += dy;
      const auto& dxhat = dy * scale;
      dmean += - dxhat * inv_stdev;
      dvar += - dxhat * (x - mean) * dvar_factor;
    }
  }
  shared_dscale[tid] = dscale;
  shared_dbias[tid] = dbias;
  shared_dmean[tid] = dmean;
  shared_dvar[tid] = dvar;

  // Compute gradients with shared memory reduction
  // @todo unroll loops
  for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
    __syncthreads();
    if (tid < stride) {
      shared_dscale[tid] += shared_dscale[tid + stride];
      shared_dbias[tid] += shared_dbias[tid + stride];
      shared_dmean[tid] += shared_dmean[tid + stride];
      shared_dvar[tid] += shared_dvar[tid + stride];
    }
  }

  // Output channel sum to global memory
  if (tid == 0) {
    cuda::atomic_add(&global_dscale[bidy], shared_dscale[0]);
    cuda::atomic_add(&global_dbias[bidy], shared_dbias[0]);
    cuda::atomic_add(&global_dmean[bidy], shared_dmean[0]);
    cuda::atomic_add(&global_dvar[bidy], shared_dvar[0]);
  }

}

/** CUDA kernel to compute gradients w.r.t. input. */
template <El::Int block_size>
__global__ void backprop2_kernel(
  El::Int channel_height,
  El::Int local_width,
  El::Int num_per_sum,
  const DataType * __restrict__ global_input,
  El::Int input_ldim,
  const DataType * __restrict__ global_gradient_wrt_output,
  El::Int gradient_wrt_output_ldim,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
  const DataType * __restrict__ global_dmean,
  const DataType * __restrict__ global_dvar,
        DataType * __restrict__ global_gradient_wrt_input,
  El::Int gradient_wrt_input_ldim) {

  // Indices
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];
  const auto& dmean = global_dmean[bidy];
  const auto& dvar = global_dvar[bidy];

  // Compute useful constants
  const auto& inv_stdev = cuda::rsqrt(var + epsilon);
  const auto& dmean_term = dmean / num_per_sum;
  const auto& dvar_term = dvar * 2 / (num_per_sum - 1);

  // Apply batch normalization
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < local_width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& dy = global_gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      const auto& dxhat = dy * scale;
      auto& dx = global_gradient_wrt_input[row + col * gradient_wrt_input_ldim];
      dx = dxhat * inv_stdev + dmean_term + dvar_term * (x - mean);
    }
  }

}

} // namespace

#ifdef LBANN_HAS_DISTCONV

template <>
void batch_normalization_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::fp_compute_distconv() {
  dc::MPIPrintStreamDebug() << get_name() << ": " << __FUNCTION__;
  assert_always(distconv_enabled());

  const bool is_training = this->m_model->get_execution_context().get_execution_mode() == execution_mode::training;

  if (keep_original_input()) {
    const auto& c = static_cast<sgd_execution_context&>(
        this->m_model->get_execution_context());
    const auto& mini_batch_size = c.get_current_mini_batch_size();
    assert_eq(mini_batch_size, get_prev_activations().Width());
  }

  assert0(dc::tensor::View(
      m_scale_t, get_weights()[0]->get_values().LockedBuffer()));
  assert0(dc::tensor::View(
      m_bias_t, get_weights()[1]->get_values().LockedBuffer()));
  assert0(dc::tensor::View(
      m_running_mean_t, get_weights()[2]->get_values().Buffer()));
  assert0(dc::tensor::View(
      m_running_var_t, get_weights()[3]->get_values().Buffer()));

  m_bn->forward_stage1(m_prev_activations_t,
                       m_mean_t,
                       m_var_t,
                       is_training, false);

  if (m_statistics_group_size == 0) {
    m_comm->allreduce(*m_mean_and_var, m_mean_and_var->RedundantComm(),
                      El::mpi::SUM);
  } else if (m_statistics_group_size == 1) {
    // Local aggregation
  } else {
    LBANN_ERROR("statics_group_size must be either 0 or 1 for now.");
  }

  m_bn->forward_stage2(m_prev_activations_t,
                       m_mean_t,
                       m_var_t,
                       m_running_mean_t,
                       m_running_var_t,
                       m_scale_t,
                       m_bias_t,
                       m_activations_t,
                       is_training);

  copy_out_activations();
}

template <>
void batch_normalization_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::bp_compute_distconv() {
  dc::MPIPrintStreamDebug() << get_name() << ": " << __FUNCTION__;
  assert_always(distconv_enabled());

  // Check execution mode
  const bool is_training = this->m_model->get_execution_context().get_execution_mode() == execution_mode::training;
  assert_always(is_training);
  const auto& c = static_cast<const sgd_execution_context&>(this->m_model->get_execution_context());
  const auto effective_mini_batch_size = c.get_effective_mini_batch_size();

  assert0(dc::tensor::View(
      m_scale_t, get_weights()[0]->get_values().LockedBuffer()));

  m_bn->backward_stage1(m_prev_activations_t,
                        m_prev_error_signals_t,
                        m_mean_t, m_var_t, m_scale_t,
                        m_scale_gradient_t, m_bias_gradient_t,
                        m_mean_gradient_t, m_var_gradient_t,
                        false);

  // Verbatim copy from bp_compute_gpu
  // Accumulate gradients
  if (is_training) {
    if (m_statistics_group_size == 0) {
      m_comm->allreduce(*m_mean_and_var_gradient,
                        m_mean_and_var_gradient->RedundantComm(),
                        El::mpi::SUM);
    }
  } else {
    Zero(*m_mean_and_var_gradient);
  }

  optimizer* scale_optimizer = m_weights[0]->get_optimizer();
  if (scale_optimizer != nullptr) {
    scale_optimizer->add_to_gradient(
        *m_scale_gradient,
        DataType(1) / effective_mini_batch_size,
        true);
  }
  optimizer* bias_optimizer = m_weights[1]->get_optimizer();
  if (bias_optimizer != nullptr) {
    bias_optimizer->add_to_gradient(
        *m_bias_gradient,
        DataType(1) / effective_mini_batch_size,
        true);
  }

  m_bn->backward_stage2(m_prev_activations_t,
                        m_prev_error_signals_t,
                        m_mean_t, m_var_t, m_scale_t,
                        m_mean_gradient_t, m_var_gradient_t,
                        m_error_signals_t);

  copy_out_error_signals();
}

#endif

template <>
void batch_normalization_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::fp_compute() {
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled()) {
    fp_compute_distconv();
    if (!early_terminate_last_iteration() || !keep_original()) {
      return;
    }
  }
#endif // LBANN_HAS_DISTCONV
  constexpr DataType one = 1;
  const bool is_training = this->m_model->get_execution_context().get_execution_mode() == execution_mode::training;

  // CUDA objects
  CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
  auto&& stream = El::GPUManager::Stream();

  // Matrices
  const auto& input = get_prev_activations();
  const auto& local_input = input.LockedMatrix();
  auto& local_output = get_local_activations();

  // Matrix parameters
  const auto& width = input.Width();
  const auto& local_width = local_input.Width();
  const auto& output_dims = get_output_dims();
  const auto& num_channels = output_dims[0];
  const auto& channel_size = get_output_size() / num_channels;

  // Compute statistics
  if (is_training) {

    // Local matrices
    auto& local_mean = m_mean_v->Matrix();
    auto& local_var = m_var_v->Matrix();
    auto& local_running_mean = this->m_weights[2]->get_values().Matrix();
    auto& local_running_var = this->m_weights[3]->get_values().Matrix();

    // Compute sums and sums of squares
    El::Zero(local_mean);
    El::Zero(local_var);
    if (!local_input.IsEmpty()) {
      const El::Int block_size = 256;
      dim3 block_dims, grid_dims;
      block_dims.x = block_size;
      grid_dims.x = (channel_size + block_size - 1) / block_size;
      grid_dims.y = num_channels;
      channel_sums_kernel<block_size>
        <<<grid_dims, block_dims, 0, stream>>>(
          channel_size, local_width,
          local_input.LockedBuffer(), local_input.LDim(),
          local_mean.Buffer(), local_var.Buffer());
    }
    El::Int num_per_sum;
    if (m_statistics_group_size == 0) {
      // Global statistics aggregation; allreduce on fused buffer.
      m_comm->allreduce(*m_mean_and_var, m_mean_and_var->RedundantComm(),
                        El::mpi::SUM);
      num_per_sum = channel_size * width;
    } else if (m_statistics_group_size == 1) {
      // Local aggregation, no allreduce needed.
      num_per_sum = channel_size * local_width;
    } else {
      // Grouped batchnorm. Allreduce on fused buffer.
      m_comm->allreduce(*m_mean_and_var,
                        m_comm->get_packed_group_comm(m_statistics_group_size),
                        El::mpi::SUM);
      if (m_num_per_sum_cache.count(width) == 0) {
        num_per_sum = channel_size * local_width;
        num_per_sum = m_comm->allreduce(
          num_per_sum, m_comm->get_packed_group_comm(m_statistics_group_size));
        m_num_per_sum_cache[width] = num_per_sum;
      } else {
        num_per_sum = m_num_per_sum_cache[width];
      }
    }

    // Compute minibatch statistics
    if (num_per_sum <= 1) {
      El::Fill(local_var, one);
    } else if (num_channels > 0) {
      const El::Int block_dim = 256;
      const El::Int grid_dim = (num_channels + block_dim - 1) / block_dim;
      compute_statistics_kernel
        <<<grid_dim, block_dim, 0, stream>>>(
          num_channels, num_per_sum, m_epsilon, m_decay,
          local_mean.Buffer(), local_var.Buffer(),
          local_running_mean.Buffer(), local_running_var.Buffer());
    }

  }

  // Apply batch normalization
  const auto& local_scale = this->m_weights[0]->get_values().LockedMatrix();
  const auto& local_bias = this->m_weights[1]->get_values().LockedMatrix();
  const auto& local_mean = (is_training ?
                            m_mean_v->LockedMatrix() :
                            this->m_weights[2]->get_values().LockedMatrix());
  const auto& local_var = (is_training ?
                           m_var_v->LockedMatrix() :
                           this->m_weights[3]->get_values().LockedMatrix());
  if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    batch_normalization_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width,
        local_input.LockedBuffer(), local_input.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), m_epsilon,
        local_scale.LockedBuffer(), local_bias.LockedBuffer(),
        local_output.Buffer(), local_output.LDim());
  }
#ifdef LBANN_HAS_DISTCONV
  dump_reference_activations();
#endif // LBANN_HAS_DISTCONV
}

template <>
void batch_normalization_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::bp_compute() {
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled()) {
    bp_compute_distconv();
    if (!early_terminate_last_iteration() || !keep_original()) {
      return;
    }
    assert0(dc::tensor::View(
        m_error_signals_copyout,
        get_error_signals().Buffer()));
    m_error_signals_copyout.zero(dc::get_stream());
  }
#endif // LBANN_HAS_DISTCONV
  constexpr DataType one = 1;
  const bool is_training = this->m_model->get_execution_context().get_execution_mode() == execution_mode::training;

  // CUDA objects
  CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
  auto&& stream = El::GPUManager::Stream();

  // Matrices
  const auto& local_scale = this->m_weights[0]->get_values().LockedMatrix();
  const auto& local_mean = (is_training ?
                            m_mean_v->LockedMatrix() :
                            this->m_weights[2]->get_values().LockedMatrix());
  const auto& local_var = (is_training ?
                           m_var_v->LockedMatrix() :
                           this->m_weights[3]->get_values().LockedMatrix());
  const auto& input = get_prev_activations();
  const auto& local_input = input.LockedMatrix();
  const auto& local_gradient_wrt_output = get_local_prev_error_signals();
  auto& local_gradient_wrt_input = get_local_error_signals();
  auto& local_mean_gradient = m_mean_gradient_v->Matrix();
  auto& local_var_gradient = m_var_gradient_v->Matrix();
  auto& local_scale_gradient = m_scale_gradient->Matrix();
  auto& local_bias_gradient = m_bias_gradient->Matrix();

  // Matrix parameters
  const auto& c = static_cast<const sgd_execution_context&>(this->m_model->get_execution_context());
  const auto effective_mini_batch_size = c.get_effective_mini_batch_size();
  const auto& width = input.Width();
  const auto& local_width = local_input.Width();
  const auto& output_dims = get_output_dims();
  const auto& num_channels = output_dims[0];
  const auto& channel_size = get_output_size() / num_channels;

  // Compute local gradients
  // Compute gradients w.r.t. batch norm parameters
  El::Zero(local_scale_gradient);
  El::Zero(local_bias_gradient);
  El::Zero(local_mean_gradient);
  El::Zero(local_var_gradient);
  if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    backprop1_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width,
        local_input.LockedBuffer(), local_input.LDim(),
        local_gradient_wrt_output.LockedBuffer(), local_gradient_wrt_output.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), m_epsilon,
        local_scale.LockedBuffer(),
        local_scale_gradient.Buffer(), local_bias_gradient.Buffer(),
        local_mean_gradient.Buffer(), local_var_gradient.Buffer());
  }

  // Accumulate gradients
  if (is_training) {
    if (m_statistics_group_size == 0) {
      // Global aggregation; allreduce on fused buffer.
      m_comm->allreduce(*m_mean_and_var_gradient,
                        m_mean_and_var_gradient->RedundantComm(),
                        El::mpi::SUM);
    } else if (m_statistics_group_size > 1) {
      // Grouped batchnorm; allreduce on fused buffer.
      m_comm->allreduce(*m_mean_and_var_gradient,
                        m_comm->get_packed_group_comm(m_statistics_group_size),
                        El::mpi::SUM);
    }
  } else {
    // Zero fused buffer.
    El::Zero(*m_mean_and_var_gradient);
  }
  optimizer* scale_optimizer = m_weights[0]->get_optimizer();
  if (scale_optimizer != nullptr) {
    scale_optimizer->add_to_gradient(*m_scale_gradient,
                                     one / effective_mini_batch_size,
                                     true);
  }
  optimizer* bias_optimizer = m_weights[1]->get_optimizer();
  if (bias_optimizer != nullptr) {
    bias_optimizer->add_to_gradient(*m_bias_gradient,
                                    one / effective_mini_batch_size,
                                    true);
  }

  // Compute error signal
  El::Int num_per_sum;
  if (m_statistics_group_size == 0) {
    // Global statistics aggregation.
    num_per_sum = channel_size * width;
  } else if (m_statistics_group_size == 1) {
    // Local aggregation.
    num_per_sum = channel_size * local_width;
  } else {
    // Grouped batchnorm.
    num_per_sum = m_num_per_sum_cache[width];  // This was computed in FP.
  }
  if (num_per_sum <= 1) {
    El::Zero(local_gradient_wrt_input);
  } else if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    backprop2_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width, num_per_sum,
        local_input.LockedBuffer(), local_input.LDim(),
        local_gradient_wrt_output.LockedBuffer(), local_gradient_wrt_output.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), m_epsilon,
        local_scale.LockedBuffer(),
        local_mean_gradient.LockedBuffer(), local_var_gradient.LockedBuffer(),
        local_gradient_wrt_input.Buffer(), local_gradient_wrt_input.LDim());
  }
#ifdef LBANN_HAS_DISTCONV
  dump_reference_error_signals();
#endif // LBANN_HAS_DISTCONV
}

template class batch_normalization_layer<
  data_layout::DATA_PARALLEL, El::Device::GPU>;

} // namespace lbann
