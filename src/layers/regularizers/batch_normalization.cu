#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2016, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#include "lbann/layers/regularizers/batch_normalization.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

// Atomic add functions
#if __CUDA_ARCH__ >= 530
__device__ inline __half atomic_add(__half* address, __half val) {
  static_cast<void>(static_cast<__half (*)(__half*, __half)>(atomic_add)); // Suppress "unused function" warning
#if 0 // TODO: replace this once Nvidia implements atomicAdd for __half
  return atomicAdd(address, val);
#else
  unsigned int* address_as_uint = (unsigned int*) address;
  unsigned int old = *address_as_uint;
  __half* old_as_half = (__half*) &old;
  unsigned int assumed;
  unsigned int updated;
  __half* updated_as_half = (__half*) &updated;
  do {
    assumed = old;
    updated = old;
    *updated_as_half += value;
    old = atomicCAS(address_as_uint, assumed, updated);
  } while (assumed != old);
  return *old_as_half;
#endif // 0
}
#endif // __CUDA_ARCH__ >= 530
__device__ inline float atomic_add(float* address, float val) {
  static_cast<void>(static_cast<float (*)(float*, float)>(atomic_add)); // Suppress "unused function" warning
  return atomicAdd(address, val);
}
__device__ inline double atomic_add(double* address, double val) {
  static_cast<void>(static_cast<double (*)(double*, double)>(atomic_add)); // Suppress "unused function" warning
#if __CUDA_ARCH__ >= 600
  return atomicAdd(address, val);
#else
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
#endif // __CUDA_ARCH__ < 600
}

// Reciprocal square root functions
#if __CUDA_ARCH__ >= 530
__device__ inline float rsqrt_(__half x) {
  static_cast<void>(static_cast<__half (*)(__half)>(rsqrt_)); // Suppress "unused function" warning
  return hrsqrt(x);
}
#endif // __CUDA_ARCH__ >= 530
__device__ inline float rsqrt_(float x) {
  static_cast<void>(static_cast<float (*)(float)>(rsqrt_)); // Suppress "unused function" warning
  return rsqrtf(x);
}
__device__ inline double rsqrt_(double x) {
  static_cast<void>(static_cast<double (*)(double)>(rsqrt_)); // Suppress "unused function" warning
  return rsqrt(x);
}

/** CUDA kernel to compute channel sums.
 *  Sums and squares of sums are used to compute mean and variance.
 */
template <El::Int block_size>
__global__ void channel_sums_kernel(
  El::Int channel_height,
  El::Int width,
  const DataType * __restrict__ data, El::Int data_ldim,
        DataType * __restrict__ sums,
        DataType * __restrict__ sqsums) {

  // Indices
  const El::Int tid = threadIdx.x;
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Initialize shared memory
  __shared__ DataType shared_sums[block_size];
  __shared__ DataType shared_sqsums[block_size];

  // Compute row sums in shared memory
  DataType private_sum = 0;
  DataType private_sqsum = 0;
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < width; ++col) {
      const auto& x = data[row + col * data_ldim];
      private_sum += x;
      private_sqsum += x * x;
    }
  }
  shared_sums[tid] = private_sum;
  shared_sqsums[tid] = private_sqsum;

  // Compute channel sum with shared memory reduction
  /// @todo unroll loops
  for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
    __syncthreads();
    if(tid < stride) {
      shared_sums[tid] += shared_sums[tid + stride];
      shared_sqsums[tid] += shared_sqsums[tid + stride];
    }
  }

  // Output channel sum to global memory
  if (tid == 0) {
    atomic_add(&sums[bidy], shared_sums[0]);
    atomic_add(&sqsums[bidy], shared_sqsums[0]);
  }

}

/** CUDA kernel to compute statistics.
 *  On input, global_mean and global_var are assumed to contain sums
 *  and squares of sums, respectively.
 */
__global__ void compute_statistics_kernel(
  El::Int num_sums,
  El::Int num_per_sum,
  DataType epsilon,
  DataType decay,
  DataType * __restrict__ global_mean,
  DataType * __restrict__ global_var,
  DataType * __restrict__ global_running_mean,
  DataType * __restrict__ global_running_var) {
  constexpr DataType one = 1;
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int i = gid; i < num_sums; i += num_threads) {

    // Compute mean and variance
    const auto& mean = global_mean[i] / num_per_sum;
    const auto& sqmean = global_var[i] / num_per_sum;
    auto var = num_per_sum * (sqmean - mean * mean) / (num_per_sum - 1);
    var = var > epsilon ? var : epsilon;
    global_mean[gid] = mean;
    global_var[gid] = var;

    // Compute running statistics
    auto& running_mean = global_running_mean[gid];
    auto& running_var = global_running_var[gid];
    running_mean = decay * running_mean + (one - decay) * mean;
    running_var = decay * running_var + (one - decay) * var;

  }

}

/** CUDA kernel to apply batch normalization. */
template <El::Int block_size>
__global__ void batch_normalization_kernel(
  El::Int channel_height,
  El::Int width,
  const DataType * __restrict__ global_input, El::Int input_ldim,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
  const DataType * __restrict__ global_bias,
        DataType * __restrict__ global_output, El::Int output_ldim) {

  // Indices
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];
  const auto& bias = global_bias[bidy];

  // Get reciprocal of standard deviation
  const auto& inv_stdev = rsqrt_(var + epsilon);

  // Apply batch normalization
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& xhat = (x - mean) * inv_stdev;
      const auto& y = scale * xhat + bias;
      global_output[row + col * output_ldim] = y;
    }
  }

}

/** CUDA kernel to compute gradients w.r.t. batch norm parameters. */
template <El::Int block_size>
__global__ void backprop1_kernel(
  El::Int channel_height,
  El::Int width,
  const DataType * __restrict__ global_input,
  El::Int input_ldim,
  const DataType * __restrict__ global_gradient_wrt_output,
  El::Int gradient_wrt_output_ldim,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
        DataType * __restrict__ global_dscale,
        DataType * __restrict__ global_dbias,
        DataType * __restrict__ global_dmean,
        DataType * __restrict__ global_dvar) {

  // Indices
  const El::Int tid = threadIdx.x;
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Initialize shared memory
  __shared__ DataType shared_dscale[block_size];
  __shared__ DataType shared_dbias[block_size];
  __shared__ DataType shared_dmean[block_size];
  __shared__ DataType shared_dvar[block_size];

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];

  // Compute useful constants
  constexpr DataType zero = 0;
  const auto& inv_stdev = rsqrt_(var + epsilon);
  const auto& dvar_factor = inv_stdev * inv_stdev * inv_stdev / 2;

  // Compute row-wise gradient contributions in shared memory
  auto dscale = zero;
  auto dbias = zero;
  auto dmean = zero;
  auto dvar = zero;
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for(El::Int col = 0; col < width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& xhat = (x - mean) * inv_stdev;
      const auto& dy = global_gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      dscale += dy * xhat;
      dbias += dy;
      const auto& dxhat = dy * scale;
      dmean += - dxhat * inv_stdev;
      dvar += - dxhat * (x - mean) * dvar_factor;
    }
  }
  shared_dscale[tid] = dscale;
  shared_dbias[tid] = dbias;
  shared_dmean[tid] = dmean;
  shared_dvar[tid] = dvar;

  // Compute gradients with shared memory reduction
  // @todo unroll loops
  for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
    __syncthreads();
    if (tid < stride) {
      shared_dscale[tid] += shared_dscale[tid + stride];
      shared_dbias[tid] += shared_dbias[tid + stride];
      shared_dmean[tid] += shared_dmean[tid + stride];
      shared_dvar[tid] += shared_dvar[tid + stride];
    }
  }

  // Output channel sum to global memory
  if (tid == 0) {
    atomic_add(&global_dscale[bidy], shared_dscale[0]);
    atomic_add(&global_dbias[bidy], shared_dbias[0]);
    atomic_add(&global_dmean[bidy], shared_dmean[0]);
    atomic_add(&global_dvar[bidy], shared_dvar[0]);
  }

}

/** CUDA kernel to compute gradients w.r.t. input. */
template <El::Int block_size>
__global__ void backprop2_kernel(
  El::Int channel_height,
  El::Int local_width,
  El::Int num_per_sum,
  const DataType * __restrict__ global_input,
  El::Int input_ldim,
  const DataType * __restrict__ global_gradient_wrt_output,
  El::Int gradient_wrt_output_ldim,
  const DataType * __restrict__ global_mean,
  const DataType * __restrict__ global_var,
  DataType epsilon,
  const DataType * __restrict__ global_scale,
  const DataType * __restrict__ global_dmean,
  const DataType * __restrict__ global_dvar,
        DataType * __restrict__ global_gradient_wrt_input,
  El::Int gradient_wrt_input_ldim) {

  // Indices
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];
  const auto& dmean = global_dmean[bidy];
  const auto& dvar = global_dvar[bidy];

  // Compute useful constants
  const auto& inv_stdev = rsqrt_(var + epsilon);
  const auto& dmean_term = dmean / num_per_sum;
  const auto& dvar_term = dvar * 2 / (num_per_sum - 1);

  // Apply batch normalization
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < local_width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& dy = global_gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      const auto& dxhat = dy * scale;
      auto& dx = global_gradient_wrt_input[row + col * gradient_wrt_input_ldim];
      dx = dxhat * inv_stdev + dmean_term + dvar_term * (x - mean);
    }
  }

}

} // namespace


#ifdef LBANN_HAS_DISTCONV

template <>
void batch_normalization_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::fp_compute_distconv() {
  dc::MPIPrintStreamDebug() << get_name() << ": " << __FUNCTION__ << "\n";
  assert_always(distconv_enabled());

  const bool is_training =
      this->m_model->get_execution_mode() == execution_mode::training;

  if (keep_original_input()) {
    assert_always(this->m_model->get_current_mini_batch_size() ==
                  get_prev_activations().Width());
  }

  assert0(dc::tensor::View(
      m_scale_t, get_weights()[0]->get_values().LockedBuffer()));
  assert0(dc::tensor::View(
      m_bias_t, get_weights()[1]->get_values().LockedBuffer()));
  assert0(dc::tensor::View(
      m_running_mean_t, get_weights()[2]->get_values().Buffer()));
  assert0(dc::tensor::View(
      m_running_var_t, get_weights()[3]->get_values().Buffer()));

  m_bn->forward(m_prev_activations_t,
                m_mean_t,
                m_var_t,
                m_running_mean_t,
                m_running_var_t,
                m_scale_t,
                m_bias_t,
                m_activations_t,
                is_training);

  copy_out_activations();
}

template <>
void batch_normalization_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::bp_compute_distconv() {
  dc::MPIPrintStreamDebug() << get_name() << ": " << __FUNCTION__ << "\n";
  assert_always(distconv_enabled());

  // Check execution mode
  const bool is_training = this->m_model->get_execution_mode() == execution_mode::training;

  //assert_always(is_training && m_use_global_stats);
  assert_always(is_training);

  assert0(dc::tensor::View(
      m_scale_t, get_weights()[0]->get_values().LockedBuffer()));

  m_bn->backward_stage1(m_prev_activations_t,
                        m_prev_error_signals_t,
                        m_mean_t, m_var_t, m_scale_t,
                        m_scale_gradient_t, m_bias_gradient_t,
                        m_mean_gradient_t, m_var_gradient_t,
                        false);

  // Verbatim copy from bp_compute_gpu
  // Accumulate gradients
  if (is_training) {
    if (m_use_global_stats) {
      m_comm->allreduce(*m_mean_gradient,
                        m_mean_gradient->RedundantComm(),
                        El::mpi::SUM);
      m_comm->allreduce(*m_var_gradient,
                        m_var_gradient->RedundantComm(),
                        El::mpi::SUM);
    }
  } else {
    Zero(*m_mean_gradient);
    Zero(*m_var_gradient);
  }

  const int effective_mini_batch_size = this->m_model->get_effective_mini_batch_size();

  optimizer* scale_optimizer = m_weights[0]->get_optimizer();
  if (scale_optimizer != nullptr) {
    scale_optimizer->add_to_gradient_staging(
        *m_scale_gradient,
        DataType(1) / effective_mini_batch_size);
  }
  optimizer* bias_optimizer = m_weights[1]->get_optimizer();
  if (bias_optimizer != nullptr) {
    bias_optimizer->add_to_gradient_staging(
        *m_bias_gradient,
        DataType(1) / effective_mini_batch_size);
  }

  m_bn->backward_stage2(m_prev_activations_t,
                        m_prev_error_signals_t,
                        m_mean_t, m_var_t, m_scale_t,
                        m_mean_gradient_t, m_var_gradient_t,
                        m_error_signals_t);

  copy_out_error_signals();
}

#endif

template <>
void batch_normalization_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::fp_compute() {
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled()) {
    fp_compute_distconv();
    if (!early_terminate_last_iteration()) {
      return;
    }
  }
#endif // LBANN_HAS_DISTCONV
  constexpr DataType one = 1;
  const bool is_training = this->m_model->get_execution_mode() == execution_mode::training;

  // CUDA objects
  CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
  auto&& stream = El::GPUManager::Stream();
  
  // Matrices
  const auto& input = get_prev_activations();
  const auto& local_input = input.LockedMatrix();
  auto& local_output = get_local_activations();

  // Matrix parameters
  const auto& width = input.Width();
  const auto& local_width = local_input.Width();
  const auto& output_dims = get_output_dims();
  const auto& num_channels = output_dims[0];
  const auto& channel_size = get_output_size() / num_channels;

  // Compute statistics
  if (is_training) {

    // Local matrices
    auto& local_mean = m_mean->Matrix();
    auto& local_var = m_var->Matrix();
    auto& local_running_mean = this->m_weights[2]->get_values().Matrix();
    auto& local_running_var = this->m_weights[3]->get_values().Matrix();

    // Compute sums and sums of squares
    El::Zero(local_mean);
    El::Zero(local_var);
    if (!local_input.IsEmpty()) {
      const El::Int block_size = 256;
      dim3 block_dims, grid_dims;
      block_dims.x = block_size;
      grid_dims.x = (channel_size + block_size - 1) / block_size;
      grid_dims.y = num_channels;
      channel_sums_kernel<block_size>
        <<<grid_dims, block_dims, 0, stream>>>(
          channel_size, local_width,
          local_input.LockedBuffer(), local_input.LDim(),
          local_mean.Buffer(), local_var.Buffer());
    }
    El::Int num_per_sum;
    if (m_use_global_stats) {
      m_comm->allreduce(*m_mean, m_mean->RedundantComm(), El::mpi::SUM);
      m_comm->allreduce(*m_var, m_var->RedundantComm(), El::mpi::SUM);
      num_per_sum = channel_size * width;
    } else {
      num_per_sum = channel_size * local_width;
    }

    // Compute minibatch statistics
    if (num_per_sum <= 1) {
      El::Fill(local_var, one);
    } else if (num_channels > 0) {
      const El::Int block_dim = 256;
      const El::Int grid_dim = (num_channels + block_dim - 1) / block_dim;
      compute_statistics_kernel
        <<<grid_dim, block_dim, 0, stream>>>(
          num_channels, num_per_sum, m_epsilon, m_decay,
          local_mean.Buffer(), local_var.Buffer(),
          local_running_mean.Buffer(), local_running_var.Buffer());
    }

  }

  // Apply batch normalization
  const auto& local_scale = this->m_weights[0]->get_values().LockedMatrix();
  const auto& local_bias = this->m_weights[1]->get_values().LockedMatrix();
  const auto& local_mean = (is_training ?
                            m_mean->LockedMatrix() :
                            this->m_weights[2]->get_values().LockedMatrix());
  const auto& local_var = (is_training ?
                           m_var->LockedMatrix() :
                           this->m_weights[3]->get_values().LockedMatrix());
  if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    batch_normalization_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width,
        local_input.LockedBuffer(), local_input.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), m_epsilon,
        local_scale.LockedBuffer(), local_bias.LockedBuffer(),
        local_output.Buffer(), local_output.LDim());
  }
#ifdef LBANN_HAS_DISTCONV
  dump_reference_activations();
#endif // LBANN_HAS_DISTCONV
}

template <>
void batch_normalization_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::bp_compute() {
#ifdef LBANN_HAS_DISTCONV
  if (distconv_enabled()) {
    bp_compute_distconv();
    if (!early_terminate_last_iteration()) {
      return;
    }
    assert0(dc::tensor::View(
        m_error_signals_copyout,
        get_error_signals().Buffer()));
    m_error_signals_copyout.zero();
  }
#endif // LBANN_HAS_DISTCONV
  constexpr DataType one = 1;
  const bool is_training = this->m_model->get_execution_mode() == execution_mode::training;

  // CUDA objects
  CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
  auto&& stream = El::GPUManager::Stream();

  // Matrices
  const auto& local_scale = this->m_weights[0]->get_values().LockedMatrix();
  const auto& local_mean = (is_training ?
                            m_mean->LockedMatrix() :
                            this->m_weights[2]->get_values().LockedMatrix());
  const auto& local_var = (is_training ?
                           m_var->LockedMatrix() :
                           this->m_weights[3]->get_values().LockedMatrix());
  const auto& input = get_prev_activations();
  const auto& local_input = input.LockedMatrix();
  const auto& local_gradient_wrt_output = get_local_prev_error_signals();
  auto& local_gradient_wrt_input = get_local_error_signals();
  auto& local_mean_gradient = m_mean_gradient->Matrix();
  auto& local_var_gradient = m_var_gradient->Matrix();
  auto& local_scale_gradient = m_scale_gradient->Matrix();
  auto& local_bias_gradient = m_bias_gradient->Matrix();

  // Matrix parameters
  const El::Int effective_mini_batch_size = this->m_model->get_effective_mini_batch_size();
  const auto& width = input.Width();
  const auto& local_width = local_input.Width();
  const auto& output_dims = get_output_dims();
  const auto& num_channels = output_dims[0];
  const auto& channel_size = get_output_size() / num_channels;

  // Compute local gradients
  // Compute gradients w.r.t. batch norm parameters
  El::Zero(local_scale_gradient);
  El::Zero(local_bias_gradient);
  El::Zero(local_mean_gradient);
  El::Zero(local_var_gradient);
  if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    backprop1_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width,
        local_input.LockedBuffer(), local_input.LDim(),
        local_gradient_wrt_output.LockedBuffer(), local_gradient_wrt_output.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), m_epsilon,
        local_scale.LockedBuffer(),
        local_scale_gradient.Buffer(), local_bias_gradient.Buffer(),
        local_mean_gradient.Buffer(), local_var_gradient.Buffer());
  }

  // Accumulate gradients
  if (is_training) {
    if (m_use_global_stats) {
      m_comm->allreduce(*m_mean_gradient,
                        m_mean_gradient->RedundantComm(),
                        El::mpi::SUM);
      m_comm->allreduce(*m_var_gradient,
                        m_var_gradient->RedundantComm(),
                        El::mpi::SUM);
    }
  } else {
    El::Zero(*m_mean_gradient);
    El::Zero(*m_var_gradient);
  }
  optimizer* scale_optimizer = m_weights[0]->get_optimizer();
  if (scale_optimizer != nullptr) {
    scale_optimizer->add_to_gradient_staging(*m_scale_gradient,
                                             one / effective_mini_batch_size);
  }
  optimizer* bias_optimizer = m_weights[1]->get_optimizer();
  if (bias_optimizer != nullptr) {
    bias_optimizer->add_to_gradient_staging(*m_bias_gradient,
                                            one / effective_mini_batch_size);
  }

  // Compute error signal
  const auto& num_per_sum = (m_use_global_stats ?
                             width * channel_size :
                             local_width * channel_size);
  if (num_per_sum <= 1) {
    El::Zero(local_gradient_wrt_input);
  } else if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    backprop2_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width, num_per_sum,
        local_input.LockedBuffer(), local_input.LDim(),
        local_gradient_wrt_output.LockedBuffer(), local_gradient_wrt_output.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), m_epsilon,
        local_scale.LockedBuffer(),
        local_mean_gradient.LockedBuffer(), local_var_gradient.LockedBuffer(),
        local_gradient_wrt_input.Buffer(), local_gradient_wrt_input.LDim());
  }
#ifdef LBANN_HAS_DISTCONV
  dump_reference_error_signals();
#endif // LBANN_HAS_DISTCONV
}

} // namespace lbann
