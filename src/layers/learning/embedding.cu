#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_EMBEDDING_LAYER_INSTANTIATE
#include "lbann/layers/learning/embedding.hpp"

namespace lbann {

namespace {

/** @brief Kernel for forward prop
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (embedding_dim / bsize) x mini_batch_size x 1
 */
__global__ void fp_kernel(El::Int num_embeddings,
                          El::Int embedding_dim,
                          El::Int mini_batch_size,
                          const DataType* __restrict__ indices,
                          El::Int indices_stride,
                          const DataType* __restrict__ embeddings,
                          El::Int embeddings_ldim,
                          DataType* __restrict__ output,
                          El::Int output_ldim) {
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const El::Int nthreadsx = blockDim.x * gridDim.x;
  const El::Int nthreadsy = blockDim.y * gridDim.y;
  for (El::Int j = gidy; j < mini_batch_size; j += nthreadsy) {
    const El::Int ind = static_cast<El::Int>(indices[j*indices_stride]);
    for (El::Int i = gidx; i < embedding_dim; i += nthreadsx) {
      auto& y = output[i+j*output_ldim];
      if (0 <= ind && ind < num_embeddings) {
        y = embeddings[i+ind*embeddings_ldim];
      }
      else {
        y = DataType{0};
      }
    }
  }
}

/** @brief Kernel for backprop
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (embedding_dim / bsize) x mini_batch_size x 1
 */
__global__ void bp_kernel(El::Int num_embeddings,
                          El::Int embedding_dim,
                          El::Int mini_batch_size,
                          El::Int padding_idx,
                          const DataType* __restrict__ indices,
                          El::Int indices_stride,
                          const DataType* __restrict__ gradient_wrt_output,
                          El::Int gradient_wrt_output_ldim,
                          DataType* __restrict__ gradient_wrt_embeddings,
                          El::Int gradient_wrt_embeddings_ldim) {
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const El::Int nthreadsx = blockDim.x * gridDim.x;
  const El::Int nthreadsy = blockDim.y * gridDim.y;
  for (El::Int j = gidy; j < mini_batch_size; j += nthreadsy) {
    const El::Int ind = static_cast<El::Int>(indices[j*indices_stride]);
    if (0 <= ind && ind < num_embeddings && ind != padding_idx) {
      for (El::Int i = gidx; i < embedding_dim; i += nthreadsx) {
        const auto& dy = gradient_wrt_output[i+j*gradient_wrt_output_ldim];
        auto& dw = gradient_wrt_embeddings[i+ind*gradient_wrt_embeddings_ldim];
        cuda::atomic_add(&dw, dy);
      }
    }
  }
}

} // namespace

template <>
void embedding_layer<data_layout::DATA_PARALLEL,El::Device::GPU>::setup_matrices(const El::Grid& grid) {
  Layer::setup_matrices(grid);
  m_gradient_wrt_embeddings.reset(new StarMat<El::Device::GPU>(grid));
}

template <>
void embedding_layer<data_layout::DATA_PARALLEL,El::Device::GPU>::fp_compute() {

  // Local data
  const auto& local_embeddings = dynamic_cast<const GPUMat&>(m_weights[0]->get_values().LockedMatrix());
  const auto& local_input = dynamic_cast<const GPUMat&>(get_local_prev_activations());
  auto& local_output = dynamic_cast<GPUMat&>(get_local_activations());

  // Launch CUDA kernel
  if (!local_input.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_output.Height() + block_size - 1) / block_size;
    grid_dims.y = local_output.Width();
    fp_kernel<<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
      m_num_embeddings,
      m_embedding_dim,
      local_input.Width(),
      local_input.LockedBuffer(),
      local_input.LDim(),
      local_embeddings.LockedBuffer(),
      local_embeddings.LDim(),
      local_output.Buffer(),
      local_output.LDim());
  }

}

template <>
void embedding_layer<data_layout::DATA_PARALLEL,El::Device::GPU>::bp_compute() {

  // Embedding layer is not differentiable w.r.t. inputs
  El::Zero(get_error_signals());

  // Nothing to be done if embeddings are not being optimized
  if (m_weights[0]->get_optimizer() == nullptr) { return; }
  auto& opt = *m_weights[0]->get_optimizer();

  // Local data
  const auto& local_input = dynamic_cast<const GPUMat&>(get_local_prev_activations());
  auto& local_embedding_grad = dynamic_cast<GPUMat&>(m_gradient_wrt_embeddings->Matrix());
  const auto& local_output_grad = dynamic_cast<const GPUMat&>(get_local_prev_error_signals());

  // Launch CUDA kernel
  El::Zero(local_embedding_grad);
  if (!local_input.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_output_grad.Height() + block_size - 1) / block_size;
    grid_dims.y = local_output_grad.Width();
    bp_kernel<<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
      m_num_embeddings,
      m_embedding_dim,
      local_input.Width(),
      m_padding_idx,
      local_input.LockedBuffer(),
      local_input.LDim(),
      local_output_grad.LockedBuffer(),
      local_output_grad.LDim(),
      local_embedding_grad.Buffer(),
      local_embedding_grad.LDim());
  }
  opt.add_to_gradient(*m_gradient_wrt_embeddings, DataType{1}, true);

}

// Explicit instantiation
template class embedding_layer<data_layout::DATA_PARALLEL, El::Device::GPU>;

} // namespace lbann
